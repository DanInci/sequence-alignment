#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "kernel_functions.h"

#define MATCH 1
#define MISMATCH -1
#define GAP -2

#define FIT_SCORE(a, b) (a == b ? MATCH : MISMATCH)

struct Alignment {
    int **score;
    char *alignedA;
    char *alignedB;
};

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"\nGPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int **needleman_wunsch_score(char *seqA, char *seqB, int lenA, int lenB) {
    char *cudaSeqA, *cudaSeqB;
    int **score, *cuda_score;

    int row_dim = lenA + 1;
    int col_dim = lenB + 1;

    printf("Allocating CUDA memory ... ");
    
    // allocate unified memory for sequences & score matrix
    gpuErrchk(hipMallocManaged(&cudaSeqA, lenA * sizeof(char)));
    gpuErrchk(hipMallocManaged(&cudaSeqB, lenB * sizeof(char)));
    gpuErrchk(hipMallocManaged(&cuda_score, row_dim * col_dim * sizeof(int)));

    gpuErrchk(hipMemcpy(cudaSeqA, seqA, lenA * sizeof(char), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(cudaSeqB, seqB, lenB * sizeof(char), hipMemcpyHostToDevice));

    printf("DONE\n");
    printf("Calculating score matrix ... ");

    cuda_compute_score<<<1, 1>>>(cudaSeqA, cudaSeqB, cuda_score, row_dim, col_dim);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    printf("DONE\n");
    printf("CUDA cleanup ... ");

    // copy score matrix from cuda memory
    score = (int **) calloc(row_dim, sizeof(int *));
    for (int i = 0; i < row_dim; i++) {
        score[i] = (int *) calloc(col_dim, sizeof(int));
        gpuErrchk(hipMemcpy(score[i], &cuda_score[i * row_dim], col_dim * sizeof(int), hipMemcpyDeviceToHost));
    }

    // deallocate cuda shared sequences & score matrix
    hipFree(cudaSeqA);
    hipFree(cudaSeqB);
    hipFree(cuda_score);

    printf("DONE\n");

    return score;
}

struct Alignment *needleman_wunsch_align(char *seqA, char *seqB, int lenA, int lenB) {
    struct Alignment *alignment;
    char *alignedA, *alignedB;
    int i, j, k;

    // allocate memory
    alignment = (struct Alignment *) calloc(1, sizeof(struct Alignment *));
    k = lenA + lenB; // maximum length of alignment is lenA + lenB
    alignedA = (char *) calloc((k + 1), sizeof(char *));
    alignedA[k] = '\0';
    alignedB = (char *) calloc((k + 1), sizeof(char *));
    alignedB[k] = '\0';
    k--;

    alignment->score = needleman_wunsch_score(seqA, seqB, lenA, lenB);

    i=lenA;
    j=lenB;

    printf("Constructing alignment ... ");

    while (i > 0 && j > 0) {
        if (alignment->score[i][j] == alignment->score[i-1][j-1] + FIT_SCORE(seqA[i-1], seqB[j-1])) {
            alignedA[k] = seqA[i-1];
            alignedB[k] = seqB[j-1];
            i--;
            j--;
        }
        else if (alignment->score[i][j] == alignment->score[i-1][j] + GAP) {
            alignedA[k] = seqA[i-1];
            alignedB[k] = '-';
            i--;
        }
        else {
            alignedA[k] = '-';
            alignedB[k] = seqB[j-1];
            j--;

        }
        k--;
    }

    while (i > 0 || j > 0) {
        if (i == 0) {
            alignedA[k] = '-';
            alignedB[k] = seqB[j-1];
            j--;
        }
        else if (j == 0) {
            alignedA[k] = seqA[i-1];
            alignedB[k] = '-';
            i--;
        }
        k--;
    }

    alignment->alignedA = alignedA+k+1;
    alignment->alignedB = alignedB+k+1;

    printf("DONE\n");

    return alignment;
}


void read_sequence(FILE *in, char **descriptor, char **seq, int *len) {
    char *line = NULL;
    size_t size = 0;
    ssize_t read; 

    if ((read = getline(&line, &size, in)) != -1) {
        line[read-1] = '\0';

        *descriptor = (char *) calloc((size + 1), sizeof(char));
        strcpy(*descriptor, line);
        
        *len = 0;
        while ((read = getline(&line, &size, in)) != -1 && memchr(line, '>', sizeof(char)) == NULL) {
            line[read-2] = '\0';

            if (*len == 0) {
                *seq = (char *) calloc(size-2, sizeof(char));
                strcpy(*seq, line);
                *len = read-2;
            }
            else {
                *seq = (char *) realloc(*seq, (*len + read) * sizeof(char));
                strcat(*seq, line);
                *len += read-2;
            }
        }

        // unread last line if it is for another sequence
        if (memchr(line, '>', sizeof(char)) != NULL) {
            fseek(in, -1 * read, SEEK_CUR);
        }

    } else {
        fprintf(stderr, "Failed to read sequence\n");
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char *argv[]) {
    char output_filename[100];
    char *seqA_descriptor, *seqA, *seqB_descriptor, *seqB;
    struct Alignment *alignment;
    FILE *input, *output;
    int lenA, lenB;

    if (argc < 2) {
        fprintf(stderr, "Usage: %s <in.seq>\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    input = fopen(argv[1], "r");
    if (input == NULL) {
        fprintf(stderr, "Failed to open input file %s\n", argv[1]);
        exit(EXIT_FAILURE);
    }

    read_sequence(input, &seqA_descriptor, &seqA, &lenA);
    read_sequence(input, &seqB_descriptor, &seqB, &lenB);

    alignment = needleman_wunsch_align(seqA, seqB, lenA, lenB);

    strcpy(output_filename, argv[1]);
    strcat(output_filename, ".cuda.aligned");
    output = fopen(output_filename, "w");
    if (output == NULL) {
        fprintf(stderr, "Failed to open output file %s\n", output_filename);
        exit(EXIT_FAILURE);
    }

    fputs(seqA_descriptor, output);
    fputc('\n', output);
    fputs(alignment->alignedA, output);
    fputc('\n', output);

    fputs(seqB_descriptor, output);
    fputc('\n', output);
    fputs(alignment->alignedB, output);
    fputc('\n', output);

    // // print score matrix
    // for (int i = 0; i <= lenA; i++) {
    //     for (int j = 0; j <= lenB; j++) {
    //         printf("%3d ", alignment->score[i][j]);
    //     }
    //     printf("\n");
    // }

    // // print sequences alignment
    // printf("\nInitial:\n%s\n%s\n\n", seqA, seqB);
    // printf("Aliniere:\n%s\n%s\n\n", alignment->alignedA, alignment->alignedB);

    // deallocate memory
    for (int i = 0; i <= lenA; i++) {
        free(alignment->score[i]);
    }
    free(alignment->score);
    free(alignment);
    free(seqA_descriptor);
    free(seqB_descriptor);
    free(seqA);
    free(seqB);


    fclose(input);
    fclose(output);

    exit(EXIT_SUCCESS);
}